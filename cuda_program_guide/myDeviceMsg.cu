#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""  
#include<iostream>  
#include <stdio.h>  
using namespace std;  
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);  
  
__global__ void addKernel(int *c, const int *a, const int *b)  
{  
    int i = threadIdx.x;  
    c[i] = a[i] + b[i];  
}  
  
int main()  
{  
    const int arraySize = 5;  
    const int a[arraySize] = { 1, 2, 3, 4, 5 };  
    const int b[arraySize] = { 10, 20, 30, 40, 50 };  
    int c[arraySize] = { 0 };  
 
    //get device prop
    hipError_t cudaStatus;
    int num = 0;
    hipDeviceProp_t deviceProp;
    cudaStatus = hipGetDeviceCount(&num);
    for(int i=0;i<num;i++)
    {
        hipGetDeviceProperties(&deviceProp, i);
        cout << "设备 " << i + 1 << " 的主要属性： " << endl;  
        cout << "设备显卡型号： " << deviceProp.name << endl; 
 
        printf("maxGridSize:%d,%d,%d\n",deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
        printf("maxThreadDim:%d,%d,%d\n",deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
        printf("warpSize:%d\n",deviceProp.warpSize);
        printf("constanMemory:%d(K)\n",deviceProp.totalConstMem/1024);

        cout << "设备全局内存总量（以MB为单位）： " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;  
        cout << "设备上一个线程块（Block）中可用的最大共享内存（以KB为单位）： " << deviceProp.sharedMemPerBlock / 1024 << endl;  
        cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << deviceProp.regsPerBlock << endl;  
        cout << "设备上一个线程块（Block）可包含的最大线程数量： " << deviceProp.maxThreadsPerBlock << endl;  
        cout << "设备的计算功能集（Compute Capability）的版本号： " << deviceProp.major << "." << deviceProp.minor << endl;  
        cout << "设备上多处理器的数量： " << deviceProp.multiProcessorCount << endl; 
    }
    return 0;  
}  
  
// Helper function for using CUDA to add vectors in parallel.  
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)  
{  
    int *dev_a = 0;  
    int *dev_b = 0;  
    int *dev_c = 0;  
    hipError_t cudaStatus;  
  
    // Choose which GPU to run on, change this on a multi-GPU system.  
    cudaStatus = hipSetDevice(0);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");  
        goto Error;  
    }  
  
    // Allocate GPU buffers for three vectors (two input, one output)    .  
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
  
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
  
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMalloc failed!");  
        goto Error;  
    }  
  
    // Copy input vectors from host memory to GPU buffers.  
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
  
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
  
    // Launch a kernel on the GPU with one thread for each element.  
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);  
  
    // hipDeviceSynchronize waits for the kernel to finish, and returns  
    // any errors encountered during the launch.  
    cudaStatus = hipDeviceSynchronize();  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);  
        goto Error;  
    }  
  
    // Copy output vector from GPU buffer to host memory.  
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);  
    if (cudaStatus != hipSuccess) {  
        fprintf(stderr, "hipMemcpy failed!");  
        goto Error;  
    }  
  
Error:  
    hipFree(dev_c);  
    hipFree(dev_a);  
    hipFree(dev_b);  
      
    return cudaStatus;  
}  
