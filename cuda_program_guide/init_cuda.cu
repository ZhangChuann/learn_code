#include"init_cuda.h"
//cuda init
bool InitCuda()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0)
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
    int i=0;
    for(i=0;i<count;i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDeviceProp(prop);
        if(hipGetDeviceProperties(&prop, i)== hipSuccess)
        {
            if(prop.major >= 1)
                break;
        }
        
    }
    if(i==count)
    {
        fprintf(stderr, "There is no device supporting CUDA\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

