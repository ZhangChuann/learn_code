#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include "../init_cuda.h"

//cuda runtime api
#include<hip/hip_runtime.h>
#define DATA_SIZE   1048576
int data[DATA_SIZE];
void GenerateNumber(int *number, int size)
{
    for(int i=0;i<size;i++)
    {
        number[i] = rand()%10;
    }
}

//global fun

__global__ void sumOfSquares(int *num, int *result, clock_t* time)
{
    int sum=0;
    printf("gpu test");
    printf("sum:%d\n", *result);
    int i;
    clock_t start = clock();
    for(i=0;i<DATA_SIZE;i++){
        printf("hh");
        sum+=num[i]*num[i]*num[i];
    }

    *result = sum;
    printf("sum gpu print:%d\n", *result);
    *time = clock() - start;
}

int main()
{
    if(!InitCuda())
        return 0;

    GenerateNumber(data, DATA_SIZE);

    int* gpudata, *result;
    clock_t *time;
    hipMalloc((void**)&gpudata, sizeof(int)*DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int));
    hipMalloc((void**)&time, sizeof(int));
    hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);
    sumOfSquares<<<1, 1, 0>>>(gpudata, result, time);
    int sum;
    clock_t time_used;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    printf("GPUsum: %d time: %d\n", sum, time_used);

    sum = 0;

    for (int i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i] * data[i];
    }

    printf("CPUsum: %d \n", sum);

    return 0;
}

