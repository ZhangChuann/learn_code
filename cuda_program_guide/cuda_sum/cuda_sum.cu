#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include "../init_cuda.h"

//cuda runtime api
#include<hip/hip_runtime.h>
#define DATA_SIZE   1048576
#define BLOCK_NUM   64
#define THREAD_NUM  256
int data[DATA_SIZE];
void GenerateNumber(int *number, int size)
{
    for(int i=0;i<size;i++)
    {
        number[i] = rand()%10;
        number[i] = 1;
    }
}

//global fun
//cycle 24949315 
__global__ void sumOfSquares(int *num, int *result, clock_t* time)
{
    int sum=0;
    printf("gpu test");
    printf("sum:%d\n", *result);
    int i;
    clock_t start = clock();
    for(i=0;i<DATA_SIZE;i++){
        sum+=num[i]*num[i]*num[i];
    }

    *result = sum;
    printf("sum gpu print:%d\n", *result);
    *time = clock() - start;
    printf("time gpu print:%d\n", *time);
}
//tree sum
//cycle 75434
__global__ void sumOfSquares_tree(int *num, int *result, clock_t* time)
{
    extern __shared__ int shared_mem[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    shared_mem[tid] = 0;
    int i=0;
    clock_t start = clock();
    if(tid==0) time[bid] = clock();

    for(i=bid*THREAD_NUM + tid; i< DATA_SIZE; i+=BLOCK_NUM*THREAD_NUM)
    {
        shared_mem[tid] += num[i]*num[i]*num[i];
    }
    __syncthreads();
    //tree sum

    int offset = 1, mask =1;

    while(offset < THREAD_NUM)
    {
        if((tid & mask) == 0)
        {
            shared_mem[tid] += shared_mem[tid + offset];
        }
        offset += offset;
        mask = offset + mask;
        __syncthreads();
    }

    if(tid == 0)
    {
        result[bid] = shared_mem[0];
        time[bid + BLOCK_NUM] = clock();
    }
}

int main()
{
    if(!InitCuda())
        return 0;

    GenerateNumber(data, DATA_SIZE);

    int* gpudata, *result;
    clock_t *time;
    hipMalloc((void**)&gpudata, sizeof(int)*DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int)*BLOCK_NUM);
    hipMalloc((void**)&time, sizeof(clock_t)*BLOCK_NUM*2);
    hipMemcpy(gpudata, data, sizeof(int)*DATA_SIZE, hipMemcpyHostToDevice);
    sumOfSquares_tree<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM*sizeof(int)>>>(gpudata, result, time);
    int sum[BLOCK_NUM];
    clock_t time_used[BLOCK_NUM*2];
    hipMemcpy(&sum, result, sizeof(int)*BLOCK_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t)*BLOCK_NUM*2, hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);

    int final_sum = 0;
    for(int i=0;i<BLOCK_NUM; i++)
    {
        final_sum += sum[i];
    }
    clock_t min_start, max_end;
    min_start = time_used[0];
    max_end = time_used[BLOCK_NUM];

    for (int i = 1; i < BLOCK_NUM; i++) {
        if (min_start > time_used[i])
            min_start = time_used[i];
        if (max_end < time_used[i + BLOCK_NUM])
            max_end = time_used[i + BLOCK_NUM];
    }

    printf("GPUsum: %d time: %d\n", final_sum, max_end - min_start);

    int cpu_sum = 0;

    for (int i = 0; i < DATA_SIZE; i++) {
        cpu_sum+= data[i] * data[i] * data[i];
    }

    printf("CPUsum: %d \n", cpu_sum);

    return 0;
}

